#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <iostream>

namespace cg = cooperative_groups;

//#define REDUCE

#define THREADS_PER_BLOCK 512
#define GRAPH_LAUNCH_ITERATIONS 100


#define NSTEP 10000
#define NKERNEL 20
#define N 500000 // tuned such that kernel takes a few microseconds

typedef struct callBackData {
  const char *fn_name;
  double *data;
} callBackData_t;


__global__ void shortKernel(float *out_d, float *in_d){
#if 0
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	  if(idx < N){
	    out_d[idx] = 1.23 * in_d[idx];
	  }
#endif
}


__global__ void reduce(float *inputVec, double *outputVec, size_t inputSize,
                       size_t outputSize) {
  __shared__ double tmp[THREADS_PER_BLOCK];

  cg::thread_block cta = cg::this_thread_block();
  size_t globaltid = blockIdx.x * blockDim.x + threadIdx.x;

  double temp_sum = 0.0;
  for (int i = globaltid; i < inputSize; i += gridDim.x * blockDim.x) {
    temp_sum += (double)inputVec[i];
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  double beta = temp_sum;
  double temp;

  for (int i = tile32.size() / 2; i > 0; i >>= 1) {
    if (tile32.thread_rank() < i) {
      temp = tmp[cta.thread_rank() + i];
      beta += temp;
      tmp[cta.thread_rank()] = beta;
    }
    cg::sync(tile32);
  }
  cg::sync(cta);

  if (cta.thread_rank() == 0 && blockIdx.x < outputSize) {
    beta = 0.0;
    for (int i = 0; i < cta.size(); i += tile32.size()) {
      beta += tmp[i];
    }
    outputVec[blockIdx.x] = beta;
  }
}

__global__ void reduceFinal(double *inputVec, double *result,
                            size_t inputSize) {
  __shared__ double tmp[THREADS_PER_BLOCK];

  cg::thread_block cta = cg::this_thread_block();
  size_t globaltid = blockIdx.x * blockDim.x + threadIdx.x;

  double temp_sum = 0.0;
  for (int i = globaltid; i < inputSize; i += gridDim.x * blockDim.x) {
    temp_sum += (double)inputVec[i];
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  // do reduction in shared mem
  if ((blockDim.x >= 512) && (cta.thread_rank() < 256)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 256];
  }

  cg::sync(cta);

  if ((blockDim.x >= 256) && (cta.thread_rank() < 128)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 128];
  }

  cg::sync(cta);

  if ((blockDim.x >= 128) && (cta.thread_rank() < 64)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 64];
  }

  cg::sync(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockDim.x >= 64) temp_sum += tmp[cta.thread_rank() + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      temp_sum += tile32.shfl_down(temp_sum, offset);
    }
  }
  // write result for this block to global mem
  if (cta.thread_rank() == 0) result[0] = temp_sum;
}

void init_input(float *a, size_t size) {
  for (size_t i = 0; i < size; i++) a[i] = (rand() & 0xFF) / (float)RAND_MAX;
}

void CUDART_CB myHostNodeCallback(void *data) {
  // Check status of GPU after stream operations are done
  callBackData_t *tmp = (callBackData_t *)(data);
  // checkCudaErrors(tmp->status);

  double *result = (double *)(tmp->data);
  char *function = (char *)(tmp->fn_name);
  printf("[%s] Host callback final reduced sum = %lf\n", function, *result);
  *result = 0.0;  // reset the result
}

void cudaGraphsManual(float *inputVec_h, float *inputVec_d, double *outputVec_d,
                      double *result_d, size_t inputSize, size_t numOfBlocks) {
  hipStream_t streamForGraph;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> nodeDependencies;
  hipGraphNode_t memcpyNode, kernelNode, memsetNode;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&streamForGraph));

  hipKernelNodeParams kernelNodeParams = {0};
  hipMemcpy3DParms memcpyParams = {0};
  hipMemsetParams memsetParams = {0};

  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr =
      make_hipPitchedPtr(inputVec_h, sizeof(float) * inputSize, inputSize, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr =
      make_hipPitchedPtr(inputVec_d, sizeof(float) * inputSize, inputSize, 1);
  memcpyParams.extent = make_hipExtent(sizeof(float) * inputSize, 1, 1);
  memcpyParams.kind = hipMemcpyHostToDevice;

  memsetParams.dst = (void *)outputVec_d;
  memsetParams.value = 0;
  memsetParams.pitch = 0;
  memsetParams.elementSize = sizeof(float);  // elementSize can be max 4 bytes
  memsetParams.width = numOfBlocks * 2;
  memsetParams.height = 1;

  checkCudaErrors(hipGraphCreate(&graph, 0));
  checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
  checkCudaErrors(
      hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams));

  nodeDependencies.push_back(memsetNode);
  nodeDependencies.push_back(memcpyNode);

  void *kernelArgs[4] = {(void *)&inputVec_d, (void *)&outputVec_d, &inputSize,
                         &numOfBlocks};

  kernelNodeParams.func = (void *)reduce;
  kernelNodeParams.gridDim = dim3(numOfBlocks, 1, 1);
  kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.kernelParams = (void **)kernelArgs;
  kernelNodeParams.extra = NULL;

  checkCudaErrors(
      hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  nodeDependencies.clear();
  nodeDependencies.push_back(kernelNode);

  memset(&memsetParams, 0, sizeof(memsetParams));
  memsetParams.dst = result_d;
  memsetParams.value = 0;
  memsetParams.elementSize = sizeof(float);
  memsetParams.width = 2;
  memsetParams.height = 1;
  checkCudaErrors(
      hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams));

  nodeDependencies.push_back(memsetNode);

  memset(&kernelNodeParams, 0, sizeof(kernelNodeParams));
  kernelNodeParams.func = (void *)reduceFinal;
  kernelNodeParams.gridDim = dim3(1, 1, 1);
  kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  void *kernelArgs2[3] = {(void *)&outputVec_d, (void *)&result_d,
                          &numOfBlocks};
  kernelNodeParams.kernelParams = kernelArgs2;
  kernelNodeParams.extra = NULL;

  checkCudaErrors(
      hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));
  nodeDependencies.clear();
  nodeDependencies.push_back(kernelNode);

  memset(&memcpyParams, 0, sizeof(memcpyParams));

  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr = make_hipPitchedPtr(result_d, sizeof(double), 1, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr = make_hipPitchedPtr(&result_h, sizeof(double), 1, 1);
  memcpyParams.extent = make_hipExtent(sizeof(double), 1, 1);
  memcpyParams.kind = hipMemcpyDeviceToHost;
  checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams));
  nodeDependencies.clear();
  nodeDependencies.push_back(memcpyNode);

  hipGraphNode_t hostNode;
  hipHostNodeParams hostParams = {0};
  hostParams.fn = myHostNodeCallback;
  callBackData_t hostFnData;
  hostFnData.data = &result_h;
  hostFnData.fn_name = "cudaGraphsManual";
  hostParams.userData = &hostFnData;

  checkCudaErrors(hipGraphAddHostNode(&hostNode, graph,
                                       nodeDependencies.data(),
                                       nodeDependencies.size(), &hostParams));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
  printf("\nNum of nodes in the graph created manually = %zu\n", numNodes);

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipGraph_t clonedGraph;
  hipGraphExec_t clonedGraphExec;
  checkCudaErrors(hipGraphClone(&clonedGraph, graph));
  checkCudaErrors(
      hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  printf("Cloned Graph Output.. \n");
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
  }
  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipGraphDestroy(clonedGraph));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
}

//limin-todo
void simpleRun(float *inputVec_h, float *inputVec_d,
               double *outputVec_d, double *result_d,
               size_t inputSize, size_t numOfBlocks) {
  hipStream_t stream1;
  //hipEvent_t forkStreamEvent, memsetEvent1, memsetEvent2;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&stream1));
#if 0
  checkCudaErrors(hipStreamCreate(&stream2));
  checkCudaErrors(hipStreamCreate(&stream3));
  checkCudaErrors(hipStreamCreate(&streamForGraph));

  checkCudaErrors(hipEventCreate(&forkStreamEvent));
  checkCudaErrors(hipEventCreate(&memsetEvent1));
  checkCudaErrors(hipEventCreate(&memsetEvent2));
#endif

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, stream1);
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
  checkCudaErrors(hipMemcpyAsync(inputVec_d, inputVec_h,
                                  sizeof(float) * inputSize, hipMemcpyDefault,
                                  stream1));
  checkCudaErrors(
      hipMemsetAsync(outputVec_d, 0, sizeof(double) * numOfBlocks, stream1));

  checkCudaErrors(hipMemsetAsync(result_d, 0, sizeof(double), stream1));

  reduce<<<numOfBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      inputVec_d, outputVec_d, inputSize, numOfBlocks);

  reduceFinal<<<1, THREADS_PER_BLOCK, 0, stream1>>>(outputVec_d, result_d,
                                                    numOfBlocks);
  checkCudaErrors(hipMemcpyAsync(&result_h, result_d, sizeof(double),
                                  hipMemcpyDefault, stream1));
 }
  hipEventRecord(e_stop, stream1); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: simpleRun time is " << elapsedTime << std::endl; 

}


//limin-todo
void simpleStreamRun(float *inputVec_h, float *inputVec_d,
               double *outputVec_d, double *result_d,
               size_t inputSize, size_t numOfBlocks) {
  hipStream_t stream1, stream2, stream3, streamForGraph;
  hipEvent_t forkStreamEvent, memsetEvent1, memsetEvent2;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&stream1));
  checkCudaErrors(hipStreamCreate(&stream2));
  checkCudaErrors(hipStreamCreate(&stream3));
  checkCudaErrors(hipStreamCreate(&streamForGraph));

  checkCudaErrors(hipEventCreate(&forkStreamEvent));
  checkCudaErrors(hipEventCreate(&memsetEvent1));
  checkCudaErrors(hipEventCreate(&memsetEvent2));

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, stream1);
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
  checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
  checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent, 0));
  checkCudaErrors(hipStreamWaitEvent(stream3, forkStreamEvent, 0));

  checkCudaErrors(hipMemcpyAsync(inputVec_d, inputVec_h,
                                  sizeof(float) * inputSize, hipMemcpyDefault,
                                  stream1));

  checkCudaErrors(
      hipMemsetAsync(outputVec_d, 0, sizeof(double) * numOfBlocks, stream2));

  checkCudaErrors(hipEventRecord(memsetEvent1, stream2));

  checkCudaErrors(hipMemsetAsync(result_d, 0, sizeof(double), stream3));
  checkCudaErrors(hipEventRecord(memsetEvent2, stream3));

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent1, 0));

  reduce<<<numOfBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      inputVec_d, outputVec_d, inputSize, numOfBlocks);

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent2, 0));

  reduceFinal<<<1, THREADS_PER_BLOCK, 0, stream1>>>(outputVec_d, result_d,
                                                    numOfBlocks);
  checkCudaErrors(hipMemcpyAsync(&result_h, result_d, sizeof(double),
                                  hipMemcpyDefault, stream1));
 }
  //hipEventRecord(e_stop, streamForGraph); 
  hipEventRecord(e_stop, stream1); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: simpleStreamRun time is " << elapsedTime << std::endl; 
 
  checkCudaErrors(hipStreamSynchronize(streamForGraph));

}



//limin-todo:
void cudaGraphsUsingStreamCapture(float *inputVec_h, float *inputVec_d,
                                  double *outputVec_d, double *result_d,
                                  size_t inputSize, size_t numOfBlocks) {
  hipStream_t stream1, stream2, stream3, streamForGraph;
  hipEvent_t forkStreamEvent, memsetEvent1, memsetEvent2;
  hipGraph_t graph;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&stream1));
  checkCudaErrors(hipStreamCreate(&stream2));
  checkCudaErrors(hipStreamCreate(&stream3));
  checkCudaErrors(hipStreamCreate(&streamForGraph));

  checkCudaErrors(hipEventCreate(&forkStreamEvent));
  checkCudaErrors(hipEventCreate(&memsetEvent1));
  checkCudaErrors(hipEventCreate(&memsetEvent2));

  checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

  checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
  checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent, 0));
  checkCudaErrors(hipStreamWaitEvent(stream3, forkStreamEvent, 0));

  checkCudaErrors(hipMemcpyAsync(inputVec_d, inputVec_h,
                                  sizeof(float) * inputSize, hipMemcpyDefault,
                                  stream1));

  checkCudaErrors(
      hipMemsetAsync(outputVec_d, 0, sizeof(double) * numOfBlocks, stream2));

  checkCudaErrors(hipEventRecord(memsetEvent1, stream2));

  checkCudaErrors(hipMemsetAsync(result_d, 0, sizeof(double), stream3));
  checkCudaErrors(hipEventRecord(memsetEvent2, stream3));

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent1, 0));

  reduce<<<numOfBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      inputVec_d, outputVec_d, inputSize, numOfBlocks);

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent2, 0));

  reduceFinal<<<1, THREADS_PER_BLOCK, 0, stream1>>>(outputVec_d, result_d,
                                                    numOfBlocks);
  checkCudaErrors(hipMemcpyAsync(&result_h, result_d, sizeof(double),
                                  hipMemcpyDefault, stream1));

  callBackData_t hostFnData = {0};
  hostFnData.data = &result_h;
  hostFnData.fn_name = "cudaGraphsUsingStreamCapture";
  hipHostFn_t fn = myHostNodeCallback;
  checkCudaErrors(hipLaunchHostFunc(stream1, fn, &hostFnData));
  checkCudaErrors(hipStreamEndCapture(stream1, &graph));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
  printf("\nNum of nodes in the graph created using stream capture API = %zu\n",
         numNodes);

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipGraph_t clonedGraph;
  hipGraphExec_t clonedGraphExec;
  checkCudaErrors(hipGraphClone(&clonedGraph, graph));
  checkCudaErrors(
      hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));
  
  hipEventRecord(e_start, 0);
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
  }
  hipEventRecord(e_stop, streamForGraph); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: cuda graph time is " << elapsedTime << std::endl; 
 
  checkCudaErrors(hipStreamSynchronize(streamForGraph));
#if 0
  printf("Cloned Graph Output.. \n");
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));
#endif

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipGraphDestroy(clonedGraph));
  checkCudaErrors(hipStreamDestroy(stream1));
  checkCudaErrors(hipStreamDestroy(stream2));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
}

int test_shortKernel_ver1(float* d_in, float* d_out){
  int threads = 512;
  int blocks = (N + threads - 1)/threads;
#if 0 
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
#endif

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, 0);
  for(int istep=0; istep<NSTEP; istep++){
	  for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
		shortKernel<<<blocks, threads, 0, 0>>>(d_out, d_in);
		hipStreamSynchronize(0);
	  }
  }
  hipEventRecord(e_stop, 0); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: shortkernel ver1 time is " << elapsedTime << std::endl; 
return 0;
}

//
int test_shortKernel_ver2(float* d_in, float* d_out){
  int threads = 512;
  int blocks = (N + threads - 1)/threads;
#if 0
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
#endif

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, 0);
  for(int istep=0; istep<NSTEP; istep++){
	  for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
		shortKernel<<<blocks, threads, 0, 0>>>(d_out, d_in);
	  }
	  //hipStreamSynchronize(0);
  }
  hipStreamSynchronize(0);
  hipEventRecord(e_stop, 0); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: shortkernel ver2 time is " << elapsedTime << std::endl; 

return 0;
}

//cuda-graph
int test_shortKernel_ver3(float* d_in, float* d_out){
  int threads = 512;
  int blocks = (N + threads - 1)/threads;

  bool graphCreated=false;
  hipGraph_t graph;
  hipGraphExec_t instance;
#if 1 
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
#endif

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, stream);
  
  for(int istep=0; istep<NSTEP; istep++){
	if(!graphCreated){
		hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
		for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
			shortKernel<<<blocks, threads, 0, stream>>>(d_out, d_in);
		}
		hipStreamEndCapture(stream, &graph);
		hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
		graphCreated=true;
	}
    hipGraphLaunch(instance, stream);
    //hipStreamSynchronize(stream);
  }

  hipEventRecord(e_stop, stream); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: shortkernel ver3 time is " << elapsedTime << std::endl; 
  
  checkCudaErrors(hipStreamDestroy(stream));
  checkCudaErrors(hipGraphExecDestroy(instance));
  checkCudaErrors(hipGraphDestroy(graph));
 return 0;
}


int test_shortKernel_ver4(float* d_in, float* d_out){
  int threads = 512;
  int blocks = (N + threads - 1)/threads;

  //bool graphCreated=false;
  hipGraph_t graph;
  hipGraphExec_t instance;
#if 1 
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
#endif
  
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
     shortKernel<<<blocks, threads, 0, stream>>>(d_out, d_in);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
  //graphCreated=true;

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, stream);
  
  for(int istep=0; istep<NSTEP; istep++){
    hipGraphLaunch(instance, stream);
    //hipStreamSynchronize(stream);
  }

  hipEventRecord(e_stop, stream); 
  hipEventSynchronize(e_stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "limin: shortkernel ver4 time (only hipGraphLaunch time) is " << elapsedTime << std::endl; 
  
  checkCudaErrors(hipStreamDestroy(stream));
  checkCudaErrors(hipGraphExecDestroy(instance));
  checkCudaErrors(hipGraphDestroy(graph));
 return 0;
}


int test_shortKernel_driver(){
  int size = N;
  float *inputVec_d = NULL, *inputVec_h = NULL;
  float *outputVec_d = NULL, *outputVec_h = NULL;

  checkCudaErrors(hipHostMalloc(&inputVec_h, sizeof(float) * size));
  checkCudaErrors(hipHostMalloc(&outputVec_h, sizeof(float) * size));
  
  checkCudaErrors(hipMalloc(&inputVec_d, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&outputVec_d, sizeof(float) * size));
  init_input(inputVec_h, size);
  
  checkCudaErrors(hipMemcpyAsync(inputVec_d, inputVec_h,
                                  sizeof(float) * size, hipMemcpyHostToDevice, 0));
                                  //sizeof(float) * size, hipMemcpyDefault, 0));
  //checkCudaErrors(hipMemsetAsync(outputVec_d, 0, sizeof(float)*size, 0));
#if 1 
  //limin-todo:
  // test_shortKernel_ver1(inputVec_d, outputVec_d);
  // test_shortKernel_ver1(inputVec_d, outputVec_d);
  //test_shortKernel_ver2(inputVec_d, outputVec_d);
  //test_shortKernel_ver3(inputVec_d, outputVec_d);
  test_shortKernel_ver4(inputVec_d, outputVec_d);
#endif

#if 1 
  checkCudaErrors(hipMemcpyAsync(outputVec_h, outputVec_d, sizeof(float)*size,
                                  hipMemcpyDeviceToHost, 0));
                                  //hipMemcpyDefault, 0));

  hipStreamSynchronize(0);
#endif

  checkCudaErrors(hipHostFree(inputVec_h));
  checkCudaErrors(hipHostFree(outputVec_h));
  
  checkCudaErrors(hipFree(inputVec_d));
  checkCudaErrors(hipFree(outputVec_d));
  
return 0;
}





int main(int argc, char **argv) {
#ifdef REDUCE
  size_t size = 1 << 24;  // number of elements to reduce
  size_t maxBlocks = 512;

  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(argc, (const char **)argv);

  printf("%zu elements\n", size);
  printf("threads per block  = %d\n", THREADS_PER_BLOCK);
  printf("Graph Launch iterations = %d\n", GRAPH_LAUNCH_ITERATIONS);

  float *inputVec_d = NULL, *inputVec_h = NULL;
  double *outputVec_d = NULL, *result_d;

  checkCudaErrors(hipHostMalloc(&inputVec_h, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&inputVec_d, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&outputVec_d, sizeof(double) * maxBlocks));
  checkCudaErrors(hipMalloc(&result_d, sizeof(double)));

  init_input(inputVec_h, size);
#if 0
  cudaGraphsManual(inputVec_h, inputVec_d, outputVec_d, result_d, size,
                   maxBlocks);
#endif
  cudaGraphsUsingStreamCapture(inputVec_h, inputVec_d, outputVec_d, result_d,
                               size, maxBlocks);
  //simpleStreamRun(inputVec_h, inputVec_d, outputVec_d, result_d, size, maxBlocks);
  //simpleRun(inputVec_h, inputVec_d, outputVec_d, result_d, size, maxBlocks);

  checkCudaErrors(hipFree(inputVec_d));
  checkCudaErrors(hipFree(outputVec_d));
  checkCudaErrors(hipFree(result_d));
  checkCudaErrors(hipHostFree(inputVec_h));
#else
  test_shortKernel_driver();
#endif
  return EXIT_SUCCESS;
}
